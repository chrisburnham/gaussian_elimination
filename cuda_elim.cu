#include "hip/hip_runtime.h"

#include "matrix_defs.h"
#include "cuda_elim.h"

__global__ cuda_sub_mul_kernel(int size,
															 floating_type* matrix,
															 floating_type* vector)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

	MATRIX_PUT(a, size, i, j, 1);
	b[i] = 2;
}

void cuda_subtract_multiples(const int size,
							 						   floating_type* matrix,
							 							 floating_type* vector)
{
	//double card_mat[size][size];
	//double card_vect[size];
	//double card_output[size];

	dim3 threads_per_block(100,100);
	dim3 num_blocks(size / threads_per_block.x, size / threads_per_block.y);
	cuda_sub_mul_kernel<<<num_blocks, threads_per_block>>>(size, matrix, vector);
}